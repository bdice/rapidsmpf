/**
 * SPDX-FileCopyrightText: Copyright (c) 2025, NVIDIA CORPORATION & AFFILIATES.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <gtest/gtest.h>
#include <thrust/equal.h>

#include <rmm/exec_policy.hpp>

#include <rapidsmpf/shuffler/chunk_batch.hpp>

// NOTE: this test is in a .cu file, because it uses thrust::equal which requires nvcc.

namespace rapidsmpf::shuffler::detail {

constexpr std::size_t operator""_KiB(unsigned long long n) {
    return n * (1 << 10);
}

// Allocate a buffer of the given size from the given resource.
std::unique_ptr<Buffer> allocate_buffer(
    MemoryType mem_type,
    std::size_t size,
    BufferResource& br,
    rmm::cuda_stream_view stream
) {
    auto [res, _] = br.reserve(mem_type, size, false);
    return br.allocate(mem_type, size, stream, res);
}

TEST(ChunkBatch, Empty) {
    uint32_t id = 1;
    Rank rank = 2;
    auto stream = rmm::cuda_stream_default;
    auto dev_mem_available = []() -> std::int64_t { return 1000_KiB; };
    BufferResource br{
        cudf::get_current_device_resource_ref(), {{MemoryType::DEVICE, dev_mem_available}}
    };

    auto test_empty_batch = [&](ChunkBatch& batch) {
        EXPECT_EQ(id, batch.id());
        EXPECT_EQ(rank, batch.destination());
        EXPECT_EQ(0, batch.size());

        // iterators won't advance
        EXPECT_EQ(batch.end(stream), batch.begin(stream));
    };

    auto batch1 = ChunkBatch::create(id, rank, {}, &br, stream);
    test_empty_batch(batch1);

    // release the metadata buffer
    auto metadata = batch1.release_metadata();
    EXPECT_EQ(ChunkBatch::batch_header_size, metadata->size());

    auto batch2 = ChunkBatch::create(std::move(metadata), {});
    test_empty_batch(batch2);

    // create chunk batch with in invalid metadata buffer -> should throw
    EXPECT_THROW(
        std::ignore = ChunkBatch::create(std::make_unique<std::vector<uint8_t>>(), {}),
        std::logic_error
    );
}

/**
 * Types of chunks
 * 1. Chunks with control messages
 * 2. Chunks with no data
 * 3. Chunks with data
 *    a. Device data
 *    b. Host data
 */

// Parametarized test for MemoryType and types of chunks
class ChunkBatchTest
    : public ::testing::TestWithParam<std::tuple<MemoryType, std::string>> {
  public:
    // dummy data for buffers
    static constexpr std::initializer_list<uint8_t> dummy_data{1, 2, 3, 4, 5, 6, 7, 8, 9};
    static constexpr std::ptrdiff_t len = dummy_data.size();

    // dummy batch details
    static constexpr uint32_t id = 1;
    static constexpr Rank rank = 2;

    // dummy data buffer
    std::unique_ptr<Buffer> data_buf;

    rmm::cuda_stream_view stream = rmm::cuda_stream_default;
    BufferResource br{cudf::get_current_device_resource_ref(), {{MemoryType::DEVICE, [] {
                                                                     return 1000_KiB;
                                                                 }}}};

    ChunkBatchTest() {
        data_buf = allocate_buffer(get_memory_type(), len, br, stream);
        // copy dummy data to the buffer
        switch (get_memory_type()) {
        case MemoryType::DEVICE:
            RAPIDSMP_CUDA_TRY_ALLOC(hipMemcpyAsync(
                data_buf->data(),
                std::data(dummy_data),
                len,
                hipMemcpyHostToDevice,
                stream
            ));
            break;
        case MemoryType::HOST:
            std::memcpy(data_buf->data(), std::data(dummy_data), len);
            break;
        }
    }

    MemoryType get_memory_type() const {
        return std::get<0>(GetParam());
    }

    auto copy_metadata() const {
        return std::make_unique<std::vector<uint8_t>>(dummy_data);
    }

    auto copy_data() const {
        return data_buf->copy_slice(0, len, stream);
    }

    auto gen_chunks() const {
        std::vector<Chunk> chunks;

        auto const& chunks_type = std::get<1>(GetParam());
        if (chunks_type == "mixed") {
            chunks.emplace_back(1, 1, 0, len, copy_metadata(), copy_data());
            chunks.emplace_back(2, 2, 0, len, copy_metadata(), copy_data());
            chunks.emplace_back(3, 3, 100);
            chunks.emplace_back(4, 4, 0, len, copy_metadata(), copy_data());
            chunks.emplace_back(5, 5, 101);
            chunks.emplace_back(6, 6, 0, len, copy_metadata(), copy_data());
        } else if (chunks_type == "no_data") {
            chunks.emplace_back(3, 3, 100);
            chunks.emplace_back(5, 5, 101);
        }
        // else -> empty chunks

        return chunks;
    }

    void test_batch(std::vector<Chunk> const& exp_chunks, ChunkBatch const& batch) const {
        EXPECT_EQ(id, batch.id());
        EXPECT_EQ(rank, batch.destination());
        EXPECT_EQ(exp_chunks.size(), batch.size());

        // std::vector<Chunk> const chunks = batch.get_chunks(stream);
        // EXPECT_EQ(exp_chunks.size(), chunks.size());

        auto chunk = batch.begin(stream);
        for (size_t i = 0; i < exp_chunks.size(); i++, chunk++) {
            SCOPED_TRACE("chunk " + std::to_string(i));
            EXPECT_EQ(exp_chunks[i].pid, chunk->pid);
            EXPECT_EQ(exp_chunks[i].cid, chunk->cid);
            EXPECT_EQ(exp_chunks[i].expected_num_chunks, chunk->expected_num_chunks);
            EXPECT_EQ(exp_chunks[i].gpu_data_size, chunk->gpu_data_size);

            if (exp_chunks[i].metadata) {
                SCOPED_TRACE("chunk metadata" + std::to_string(i));
                EXPECT_EQ(*exp_chunks[i].metadata, *chunk->metadata);
            }

            if (exp_chunks[i].gpu_data) {
                switch (get_memory_type()) {
                case MemoryType::DEVICE:
                    {
                        SCOPED_TRACE("chunk data device" + std::to_string(i));
                        EXPECT_TRUE(
                            thrust::equal(
                                rmm::exec_policy(stream, br.device_mr()),
                                static_cast<cuda::std::byte*>(
                                    exp_chunks[i].gpu_data->data()
                                ),
                                static_cast<cuda::std::byte*>(
                                    exp_chunks[i].gpu_data->data()
                                ) + len,
                                static_cast<cuda::std::byte*>(chunk->gpu_data->data())
                            )
                        );
                        break;
                    }
                case MemoryType::HOST:
                    {
                        SCOPED_TRACE("chunk data host" + std::to_string(i));
                        EXPECT_EQ(
                            *(const_cast<Buffer const&>(*exp_chunks[i].gpu_data).host()),
                            *(const_cast<Buffer const&>(*chunk->gpu_data).host())
                        );
                        break;
                    }
                }
            }
        }
        EXPECT_EQ(batch.end(stream), chunk);  // check if the iterator has reached the end
    };
};

INSTANTIATE_TEST_SUITE_P(
    ChunkBatchTestP,
    ChunkBatchTest,
    ::testing::Combine(
        ::testing::Values(MemoryType::DEVICE, MemoryType::HOST),
        ::testing::Values("mixed", "no_data", "empty")
    ),
    [](const ::testing::TestParamInfo<ChunkBatchTest::ParamType>& info) {
        return std::string(
                   std::get<0>(info.param) == MemoryType::HOST ? "host" : "device"
               )
               + "__" + std::get<1>(info.param);
    }
);

TEST_P(ChunkBatchTest, Run) {
    auto exp_chunks = gen_chunks();

    auto batch1 = ChunkBatch::create(id, rank, gen_chunks(), &br, stream);
    EXPECT_NO_FATAL_FAILURE(test_batch(exp_chunks, batch1));

    // reacreate a new batch using the first batch
    auto batch2 = ChunkBatch::create(batch1.release_metadata(), batch1.release_payload());
    EXPECT_NO_FATAL_FAILURE(test_batch(exp_chunks, batch2));
}

}  // namespace rapidsmpf::shuffler::detail
